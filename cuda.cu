#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 10000000


static int ones[N];
static int negativeOnes[N];
static int zeros[N];
static int sum[N];

__global__ void vector_add(int *sum, int *a, int *b, int n) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for(int i = index; i < n; i += stride){
        sum[i] = a[i] + b[i];
    }
}

void setUp(int *arr, int val)
{
    for (int i = 0; i < N; i++)
    {
        arr[i] = val;
    }
}

int main()
{
    int size = N * sizeof(int);
    setUp(ones, 1);
    setUp(negativeOnes, -1);
    setUp(zeros, 0);

    int *cudaOnes, *cudaNegativeOnes, *cudaSum;
    hipMalloc((void**)&cudaOnes, size);
    hipMalloc((void**)&cudaNegativeOnes, size);
    hipMalloc((void**)&cudaSum, size);

    hipMemcpy(cudaOnes, ones, size, hipMemcpyHostToDevice);
    hipMemcpy(cudaNegativeOnes, negativeOnes, size, hipMemcpyHostToDevice);

    
    vector_add<<<1, 256>>>(cudaSum, cudaOnes, cudaNegativeOnes,N);
    
    hipMemcpy(sum, cudaSum, size, hipMemcpyDeviceToHost);

    hipFree(cudaNegativeOnes);
    hipFree(cudaOnes);
    hipFree(cudaSum);
    
    for (int i = 0; i < N; i++)
    {
        if (sum[i] != zeros[i])
        {
            printf("Not Summed Correctly");
        }
    }

    return 0;
}